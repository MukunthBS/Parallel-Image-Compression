#include "hip/hip_runtime.h"
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <climits>

#include "../include/config_cuda.hh"

__constant__ int cudaQuantArr[WINDOW_Y * WINDOW_X];
__constant__ float cudaCosArr1[WINDOW_Y * WINDOW_X];
__constant__ float cudaCosArr2[WINDOW_Y * WINDOW_X];
__constant__ float cudaOne_by_root_2;
__constant__ float cudaOne_by_root_2N;
__constant__ float cudaTerm3;
__constant__ float cudaTerm4;
uint8_t *cudaImg;


__device__
void discreteCosTransformCuda(const int *grayData, float *patchDCT, int offset, const int &linearIdx) {
    float cos1, cos2, temp;
    int tx = threadIdx.x % WINDOW_X;
    int ty = threadIdx.y % WINDOW_Y;
    int offsetX = (int)(threadIdx.x / WINDOW_X) * WINDOW_X;
    int offsetY = (int)(threadIdx.y / WINDOW_Y) * WINDOW_Y;
    int x, y;
    temp = 0.0;
    // Loop over all pixels in patch
    for (y = 0; y < WINDOW_Y; y++) {
        for (x = 0; x < WINDOW_X; x++) {
            cos1 = cudaCosArr1[y * WINDOW_X + ty];
            cos2 = cudaCosArr2[x * WINDOW_X + tx];
            temp += grayData[(y + offsetY) * blockDim.x + (x + offsetX)] * cos1 * cos2;
        }
    }

    temp *= cudaOne_by_root_2N;
    if (ty > 0) {
        temp *= cudaOne_by_root_2;
    }

    if (tx > 0) {
        temp *= cudaOne_by_root_2;
    }

    patchDCT[linearIdx] = (int)temp;
}


__device__ __inline__
void quantizeCuda(const float *patchDCT, int *quantData, const int &linearIdx) {
    int tx = threadIdx.x % WINDOW_X;
    int ty = threadIdx.y % WINDOW_Y;
    quantData[linearIdx] = (int)roundf((float)patchDCT[linearIdx]
                                       / cudaQuantArr[ty * WINDOW_Y + tx]);
}


__device__ __inline__
void dequantizeCuda(const int *quantData, int *dequantData, const int &linearIdx) {
    int tx = threadIdx.x % WINDOW_X;
    int ty = threadIdx.y % WINDOW_Y;
    dequantData[linearIdx] = quantData[linearIdx] * cudaQuantArr[ty * WINDOW_Y + tx];
}


__device__
void invDiscreteCosTransformCuda(const int *dequantData, int *patchInverseDCT, int offset, const int &linearIdx) {
    int x, y;
    float cos1, cos2, temp;
    int tx = threadIdx.x % WINDOW_X;
    int ty = threadIdx.y % WINDOW_Y;
    int offsetX = (int)(threadIdx.x / WINDOW_X) * WINDOW_X;
    int offsetY = (int)(threadIdx.y / WINDOW_Y) * WINDOW_Y;

    // 1st value
    temp = 1/4. * (float)dequantData[(0 + offsetY) * blockDim.x + (0 + offsetX)];
    // First column values
    for (y = 1; y < WINDOW_Y; y++) {
        temp += 1/2. * (float)dequantData[(y + offsetY) * blockDim.x + (0 + offsetX)];
    }
    // First row values
    for (x = 1; x < WINDOW_X; x++) {
        temp += 1/2. * (float)dequantData[(0 + offsetY) * blockDim.x + (x + offsetX)];
    }
    // Loop over all pixels in patch
    for (y = 1; y < WINDOW_Y; y++) {
        for (x = 1; x < WINDOW_X; x++) {
            cos1 = cudaCosArr1[y * WINDOW_X + ty];
            cos2 = cudaCosArr2[x * WINDOW_X + tx];
            temp += (float)dequantData[(y + offsetY) * blockDim.x + (x + offsetX)] * cos1 * cos2;
        }
    }

    patchInverseDCT[linearIdx] = temp * cudaTerm3 * cudaTerm4;
}


__device__ __inline__
int getOffset(int width, int i, int j) {
    /**
     *  width: image width
     *  i: pixel row
     *  j: pixel column
     */
    return (i * width + j) * NUM_CHANNELS;
}


__global__
void compressCuda(uint8_t *cudaImg, int width, int height) {
    __shared__ int grayData[BLOCKSIZE];
    __shared__ float patchDCT[BLOCKSIZE];
    __shared__ int quantData[BLOCKSIZE];
    __shared__ int dequantData[BLOCKSIZE];
    __shared__ int patchInverseDCT[BLOCKSIZE];

    int add_rows = (PIXEL - (height % PIXEL) != PIXEL ? PIXEL - (height % PIXEL) : 0);
    int add_columns = (PIXEL - (width % PIXEL) != PIXEL ? PIXEL - (width % PIXEL) : 0);

    // padded dimensions to make multiples of patch size
    int _height = height + add_rows;
    int _width = width + add_columns;

    int blockMinX = blockIdx.x * blockDim.x;
    int blockMaxX = blockMinX + blockDim.x;
    int blockMinY = blockIdx.y * blockDim.y;
    int blockMaxY = blockMinY + blockDim.y;

    blockMaxX = min(blockMaxX, _width);
    blockMaxY = min(blockMaxY, _height);

    int pixelX = blockMinX + threadIdx.x;
    int pixelY = blockMinY + threadIdx.y;

    int linearIdx = threadIdx.y * blockDim.x + threadIdx.x;
    int offset = getOffset(width, pixelY, pixelX);

    // Write grayscale data in `grayData` along with zero padding
    if (pixelX < width && pixelY < height) {
        uint8_t *bgrPixel = (uint8_t *) &cudaImg[offset];
        grayData[linearIdx] = (bgrPixel[0] + bgrPixel[1] + bgrPixel[2]) / 3.f;
    } else if (pixelX >= width) {
        grayData[linearIdx] = 0;
    } else if (pixelY >= height) {
        grayData[linearIdx] = 0;
    }

    __syncthreads();

    discreteCosTransformCuda(grayData, patchDCT, offset, linearIdx);
    quantizeCuda(patchDCT, quantData, linearIdx);
    dequantizeCuda(quantData, dequantData, linearIdx);

    __syncthreads();

    invDiscreteCosTransformCuda(dequantData, patchInverseDCT, offset, linearIdx);

    __syncthreads();

    if (pixelX >= width || pixelY >= height) {
        return;
    }

    uint8_t pixelValue = patchInverseDCT[linearIdx];
    cudaImg[offset + 0] = pixelValue;
    cudaImg[offset + 1] = pixelValue;
    cudaImg[offset + 2] = pixelValue;
}


void cudaSetup(uint8_t *img, int width, int height) {
    /* Allocate data structure for storing the image on device global memory */
    size_t num = NUM_CHANNELS * width * height;
    hipMalloc(&cudaImg, sizeof(uint8_t) * num);
    hipMemcpy(cudaImg, img, sizeof(uint8_t) * num, hipMemcpyHostToDevice);

    /* Store constants in the device global read-only memory */
    int quantArr[WINDOW_Y * WINDOW_X] = {16, 11, 12, 14, 12, 10, 16, 14,
                                         13, 14, 18, 17, 16, 19, 24, 40,
                                         26, 24, 22, 22, 24, 49, 35, 37,
                                         29, 40, 58, 51, 61, 60, 57, 51,
                                         56, 55, 64, 72, 92, 78, 64, 68,
                                         87, 69, 55, 56, 80, 109, 81, 87,
                                         95, 98, 103, 104, 103, 62, 77, 113,
                                         121, 112, 100, 120, 92, 101, 103, 99};
    float cosArr1[WINDOW_Y * WINDOW_X];
    float cosArr2[WINDOW_Y * WINDOW_X];
    for (int i = 0; i < WINDOW_Y; i++) {
        for (int j = 0; j < WINDOW_X; j++) {
            cosArr1[i * WINDOW_X + j] = cos(term1 * (i + 0.5) * j);
            cosArr2[i * WINDOW_X + j] = cos(term2 * (i + 0.5) * j);
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(cudaQuantArr), &quantArr, sizeof(int) * WINDOW_X * WINDOW_Y);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaCosArr1), &cosArr1, sizeof(float) * WINDOW_X * WINDOW_Y);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaCosArr2), &cosArr2, sizeof(float) * WINDOW_X * WINDOW_Y);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaOne_by_root_2), &one_by_root_2, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(cudaOne_by_root_2N), &one_by_root_2N, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(cudaTerm3), &term3, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(cudaTerm4), &term4, sizeof(float));
}


void compress(int width, int height) {
    // TODO: Number of rows and cols should be based on the padded dimensions.
    // Or not?
    int rows = (height + BLK_HEIGHT - 1) / BLK_HEIGHT;
    int cols = (width + BLK_WIDTH - 1) / BLK_WIDTH;
    dim3 blockDim(BLK_WIDTH, BLK_HEIGHT);
    dim3 gridDim(cols, rows);
    compressCuda<<<gridDim, blockDim>>>(cudaImg, width, height);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
}


void cudaFinish(uint8_t *img, int width, int height) {
    size_t num = NUM_CHANNELS * width * height;
    hipMemcpy(img, cudaImg, sizeof(uint8_t) * num, hipMemcpyDeviceToHost);
    hipFree(cudaImg);
}
